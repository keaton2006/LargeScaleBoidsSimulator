#include "hip/hip_runtime.h"
/*
 * LargeBoidsSimulator
 *
 * Yhoichi Mototake
 */
//#include "./book.h"
#include "gpus_controller.h"
#include <stdio.h>
#include <stdlib.h>
#include "./book.h"

#if DRAW_CUDA == 2
#include <pthread.h>
#endif
int calc_main( int argc, char **argv ) {


#if DRAW_CUDA == 1
    int deviceCount;
    HANDLE_ERROR( hipGetDeviceCount( &deviceCount ) );
    if (deviceCount < 2) {
        printf( "We need at least two compute 1.0 or greater "
                "devices, but only found %d\n", deviceCount );
        return 0;
    }

    hipDeviceProp_t  prop;
    for (int i=0; i<2; i++) {
        HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
        if (prop.canMapHostMemory != 1) {
            printf( "Device %d can not map memory.\n", i );
            return 0;
        }
    }
#else
#endif

    float4 *a;
    float3 *b;

    HANDLE_ERROR( hipSetDeviceFlags( hipDeviceMapHost ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&a, N*sizeof(float4),
                              hipHostMallocWriteCombined |
                              hipHostMallocPortable |
                              hipHostMallocMapped ) );
    HANDLE_ERROR( hipHostAlloc( (void**)&b, N*sizeof(float3),
                              hipHostMallocWriteCombined |
                              hipHostMallocPortable      |
                              hipHostMallocMapped ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i].x = i;
        b[i].y = i;
    }

    // preparation for multithread
    DataStruct  data[2];
    data[0].deviceID = 0;
    data[0].offset = 0;
    data[0].size = N/2;
    data[0].a = a;
    data[0].b = b;

    data[1].deviceID = 1;
    data[1].offset = N/2;
    data[1].size = N/2;
    data[1].a = a;
    data[1].b = b;
    data[1].pArgc = &argc;
    data[1].pArgv = argv;
    data[1].time = 0;
#if DRAW_CUDA == 1
    HANDLE_ERROR( hipSetDevice( 1 ) );
    CUTThread   thread = start_thread( rundraw, &(data[1]) );
#else
    pthread_t th_draw;
    pthread_create(&th_draw,NULL,draw,&(data[1]));
#endif

#if DRAW_CUDA == 1
    HANDLE_ERROR( hipSetDevice( 0 ) );
#endif
    routine( &(data[1]) );

    printf("101\n");
#if DRAW_CUDA == 1
    end_thread( thread );
#endif
    // free memory on the CPU side
    HANDLE_ERROR( hipHostFree( a ) );
    HANDLE_ERROR( hipHostFree( b ) );

    hipDeviceReset();
    printf("finish all\n");
    return 0;
}
